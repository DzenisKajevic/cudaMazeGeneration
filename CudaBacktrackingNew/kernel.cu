#include "hip/hip_runtime.h"
﻿#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define N 9  // Size of individual mazes (N x N)
#define P 2  // Number of mazes in one row/column of the large maze
#define MAX_SIZE (N * N)

// (N*P)*(N*P)

#define cudaCheckError() {                               \
    hipError_t e = hipGetLastError();                    \
    if (e != hipSuccess) {                                \
        printf("CUDA Error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                                \
    }                                                     \
}


enum MAZE_PATH {
    EMPTY = 0x0,
    WALL = 0x1,
    EXIT = 0x2,
    SOLUTION = 0x3,
    START = 0x4,
    PARTICLE = 0x5,
};

// Declaration (if defined later or in another file)
__device__ void initialize_maze_cuda(MAZE_PATH* maze, int size, int* exit_row, int* exit_col, hiprandState* localState);
__global__ void init_rng(hiprandState* state, unsigned long seed);
__global__ void generate_mazes(hiprandState* globalState, MAZE_PATH* mazes);
__device__ void generate_paths_cuda(MAZE_PATH* maze, int size, int* exit_coords, hiprandState* localState);
__device__ void visit_forward_cuda(MAZE_PATH* maze, int size, int& curr_index, int* curr_cell, int* curr_track, bool* visited_cells, hiprandState* localState, bool is_exit);
__device__ void get_unvisited_near_cells_cuda(MAZE_PATH* maze, int* curr_cell, int size, bool* visited_cells, int& n_cells, bool is_exit, int* near_cells);
__device__ void backtrack_cuda(MAZE_PATH* maze, int size, int& curr_index, int* curr_cell, int* curr_track, bool* visited_cells, hiprandState* localState);

void combine_mazes(MAZE_PATH* mazes, int n, int p, MAZE_PATH* large_maze);
void print_maze(MAZE_PATH* maze, int size);

// Function to print the maze
void print_maze(MAZE_PATH* maze, int size) {
    for (int row = 0; row < size; ++row) {
        for (int col = 0; col < size; ++col) {
            switch (maze[row * size + col]) {
            case MAZE_PATH::EMPTY:
                std::cout << " ";
                break;
            case MAZE_PATH::WALL:
                std::cout << "#";
                break;
            case MAZE_PATH::EXIT:
                std::cout << "E";
                break;
            case MAZE_PATH::SOLUTION:
                std::cout << ".";
                break;
            case MAZE_PATH::START:
                std::cout << "S";
                break;
            case MAZE_PATH::PARTICLE:
                std::cout << "P";
                break;
            default:
                std::cout << "?";
                break;
            }
        }
        std::cout << std::endl;
    }
}

void print_maze_debug(MAZE_PATH* maze, int size) {
    for (int i = 0; i < size * size; ++i) {
        std::cout << static_cast<int>(maze[i]) << " ";
        if ((i + 1) % size == 0) std::cout << std::endl;
    }
}



// Kernel function to initialize random states
__global__ void init_rng(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// GPU function to initialize the maze (similar to initialize_maze)
__device__ void initialize_maze_cuda(MAZE_PATH* maze, int size, int* exit_row, int* exit_col, hiprandState* localState) {
    for (int i = 0; i < size * size; ++i) {
        int row = i / size;
        int col = i % size;
        if (row % 2 == 0 || col % 2 == 0) {
            maze[i] = MAZE_PATH::WALL;
        }
        else {
            maze[i] = MAZE_PATH::EMPTY;
        }
    }

    // Randomly choose a border for the exit (0: top/bottom, 1: left/right)
    int border_choice = hiprand(localState) % 2;

    //int exit_row, exit_col;

    if (border_choice == 0) {
        // Top or bottom border
        *exit_row = (hiprand(localState) % 2 == 0) ? 0 : size - 1;
        *exit_col = (hiprand(localState) % (size / 2)) * 2 + 1;
    }
    else {
        // Left or right border
        *exit_row = (hiprand(localState) % (size / 2)) * 2 + 1;
        *exit_col = (hiprand(localState) % 2 == 0) ? 0 : size - 1;
    }
    // Ensure the exit is within bounds (mozda bude trebalo)
    //if (exit_row < size && exit_col < size) {
    //    maze[exit_row * size + exit_col] = MAZE_PATH::EXIT;
    //}
    
    maze[*exit_row * size + *exit_col] = MAZE_PATH::EXIT;
}

// Kernel function to generate individual mazes
__global__ void generate_mazes(hiprandState* globalState, MAZE_PATH* mazes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Get the random state for this thread
    hiprandState localState = globalState[idx];

    // Pointer to this thread's maze in the global memory
    MAZE_PATH* maze = &mazes[idx * N * N];

    int exit_row, exit_col;

    // Maze generation logic (pseudo-code)



    // Initialize the maze
    initialize_maze_cuda(maze, N, &exit_row, &exit_col, &localState);

    // Generate the paths
    generate_paths_cuda(maze, N, new int[2]{ exit_row, exit_col }, &localState);

    // Store the random state back to global memory
    globalState[idx] = localState;
}

// Function to combine mazes into one large maze on the CPU
void combine_mazes(MAZE_PATH* mazes, int n, int p, MAZE_PATH* large_maze) {
    int large_size = n * p;

    for (int i = 0; i < p; ++i) {
        for (int j = 0; j < p; ++j) {
            for (int x = 0; x < n; ++x) {
                for (int y = 0; y < n; ++y) {
                    large_maze[(i * n + x) * large_size + (j * n + y)] = mazes[(i * p + j) * n * n + x * n + y];
                }
            }
        }
    }
}

__device__ void generate_paths_cuda(MAZE_PATH* maze, int size, int* exit_coords, hiprandState* localState) {
    bool* visited_cells = new bool[MAX_SIZE];  // Dynamically allocate
    int* curr_track = new int[MAX_SIZE * 2];   // Dynamically allocate
    int curr_index = 0;

    int curr_cell[2] = { exit_coords[0], exit_coords[1] };
    visited_cells[curr_cell[0] * size + curr_cell[1]] = true;
    curr_track[curr_index * 2] = curr_cell[0];
    curr_track[curr_index * 2 + 1] = curr_cell[1];

    visit_forward_cuda(maze, size, curr_index, curr_cell, curr_track, visited_cells, localState, true);

    delete[] visited_cells;
    delete[] curr_track;
}

__device__ void visit_forward_cuda(MAZE_PATH* maze, int size, int& curr_index, int* curr_cell, int* curr_track, bool* visited_cells, hiprandState* localState, bool is_exit) {
    int n_cells = 0;
    int near_cells[4 * 2];  // Up to 4 possible cells, each with x and y coordinates

    get_unvisited_near_cells_cuda(maze, curr_cell, size, visited_cells, n_cells, is_exit, near_cells);
    while (n_cells > 0) {
        curr_index++;
        int new_cell_index = hiprand(localState) % n_cells;
        int* new_cell = &near_cells[new_cell_index * 2];

        if (!is_exit) {
            int row_to_del = (new_cell[0] + curr_cell[0]) / 2;
            int col_to_del = (new_cell[1] + curr_cell[1]) / 2;

            maze[row_to_del * size + col_to_del] = MAZE_PATH::EMPTY;
        }
        else {
            is_exit = false;
        }

        curr_cell[0] = new_cell[0];
        curr_cell[1] = new_cell[1];
        visited_cells[curr_cell[0] * size + curr_cell[1]] = true;
        curr_track[curr_index * 2] = curr_cell[0];
        curr_track[curr_index * 2 + 1] = curr_cell[1];

        get_unvisited_near_cells_cuda(maze, curr_cell, size, visited_cells, n_cells, is_exit, near_cells);
    }
    backtrack_cuda(maze, size, curr_index, curr_cell, curr_track, visited_cells, localState);
}

__device__ void get_unvisited_near_cells_cuda(MAZE_PATH* maze, int* curr_cell, int size, bool* visited_cells, int& n_cells, bool is_exit, int* near_cells) {
    n_cells = 0;
    int index_offset = is_exit ? 1 : 2;

    // East cell
    int index = curr_cell[0] + index_offset;
    if (index < size && !visited_cells[index * size + curr_cell[1]]) {
        near_cells[n_cells * 2] = index;
        near_cells[n_cells * 2 + 1] = curr_cell[1];
        n_cells++;
    }

    // North cell
    index = curr_cell[1] + index_offset;
    if (index < size && !visited_cells[curr_cell[0] * size + index]) {
        near_cells[n_cells * 2] = curr_cell[0];
        near_cells[n_cells * 2 + 1] = index;
        n_cells++;
    }

    // West cell
    index = curr_cell[0] - index_offset;
    if (index >= 0 && !visited_cells[index * size + curr_cell[1]]) {
        near_cells[n_cells * 2] = index;
        near_cells[n_cells * 2 + 1] = curr_cell[1];
        n_cells++;
    }

    // South cell
    index = curr_cell[1] - index_offset;
    if (index >= 0 && !visited_cells[curr_cell[0] * size + index]) {
        near_cells[n_cells * 2] = curr_cell[0];
        near_cells[n_cells * 2 + 1] = index;
        n_cells++;
    }
}

__device__ void backtrack_cuda(MAZE_PATH* maze, int size, int& curr_index, int* curr_cell, int* curr_track, bool* visited_cells, hiprandState* localState) {
    int n_cells = 0;

    for (int index = curr_index - 1; index > 0; index--) {
        curr_cell[0] = curr_track[index * 2];
        curr_cell[1] = curr_track[index * 2 + 1];
        curr_index = index;

        get_unvisited_near_cells_cuda(maze, curr_cell, size, visited_cells, n_cells, false, curr_track);
        if (n_cells > 0) {
            visit_forward_cuda(maze, size, curr_index, curr_cell, curr_track, visited_cells, localState, false);
            break;
        }
    }
}



int main() {
    int num_mazes = P * P;
    int maze_size = N * N;
    size_t mazes_bytes = num_mazes * maze_size * sizeof(MAZE_PATH);

    // Allocate memory for the mazes on the GPU
    MAZE_PATH* d_mazes;
    hipMalloc(&d_mazes, mazes_bytes);

    // Allocate memory for RNG states on the GPU
    hiprandState* d_states;
    hipMalloc(&d_states, num_mazes * sizeof(hiprandState));

    // Initialize the RNG states
    init_rng << <P, P >> > (d_states, time(NULL));
    cudaCheckError();

    hipDeviceSynchronize();

    // Generate the mazes
    generate_mazes << <P, P >> > (d_states, d_mazes);
    cudaCheckError();

    hipDeviceSynchronize();

    // Allocate memory for one maze on the CPU
    MAZE_PATH* h_single_maze = new MAZE_PATH[maze_size];

    // Copy and print each maze
    for (int i = 0; i < num_mazes; ++i) {
        // Copy the maze from the GPU to the CPU
        hipMemcpy(h_single_maze, d_mazes + i * maze_size, maze_size * sizeof(MAZE_PATH), hipMemcpyDeviceToHost);
        //cudaCheckError();

        // Print the maze
        print_maze(h_single_maze, N);

        //print_maze_debug(h_single_maze, N);

        // Optionally, add a separator between mazes
        std::cout << "--------------------\n";
    }

    //// Copy one maze from the GPU to the CPU (e.g., the first maze)
    //hipMemcpy(h_single_maze, d_mazes, maze_size * sizeof(MAZE_PATH), hipMemcpyDeviceToHost);
    //
    //// Print the maze
    //print_maze(h_single_maze, N);

    // Allocate memory for the large maze on the CPU
    MAZE_PATH* large_maze = new MAZE_PATH[N * P * N * P];

    // Combine the mazes into one large maze
    MAZE_PATH* h_mazes = new MAZE_PATH[maze_size * num_mazes];
    hipMemcpy(h_mazes, d_mazes, mazes_bytes, hipMemcpyDeviceToHost);
    combine_mazes(h_mazes, N, P, large_maze);


    // Free resources
    hipFree(d_mazes);
    hipFree(d_states);
    delete[] h_mazes;
    delete[] large_maze;

    return 0;
}